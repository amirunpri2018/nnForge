#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2017 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "lerror_layer_updater_cuda.h"

#include <hip/hip_runtime.h>

#include "util_cuda.h"
#include "../lerror_layer.h"
#include "../neural_network_exception.h"

namespace nnforge
{
	namespace cuda
	{
		extern __shared__ float arr_sh[];
		template<int n_type>
		__global__ void lerror_upd_kernel(
			float * __restrict output,
			const float * __restrict input0,
			const float * __restrict input1,
			const float * __restrict scale_mask,
			int input_feature_map_count,
			int elem_count_per_feature_map,
			float n_value,
			float scale,
			int entry_count)
		{
			int feature_map_id = threadIdx.x;
			int neuron_id = blockIdx.x;
			int entry_id = blockIdx.y;
			int threadblock_size = blockDim.x;

			float err = 0.0F;

			int output_offset = entry_id * elem_count_per_feature_map + neuron_id;

			float mask = 1.0F;
			if (scale_mask)
				mask = scale_mask[output_offset];

			int thread_id = threadIdx.x;
			if (mask != 0.0F)
			{
				int input_offset = (entry_id * input_feature_map_count + feature_map_id) * elem_count_per_feature_map + neuron_id;
				while (feature_map_id < input_feature_map_count)
				{
					float local_err = input0[input_offset] - input1[input_offset];

					if (n_type == 1)
						err += fabsf(local_err);
					else if (n_type == 2)
						err += local_err * local_err;
					else
						err += __powf(fabsf(local_err), n_value);

					feature_map_id += threadblock_size;
					input_offset += threadblock_size * elem_count_per_feature_map;
				}

				int lane_id = thread_id & 31;
				#pragma unroll
				for(int tx = 16; tx > 0; tx >>= 1)
					#ifdef __CUDACC_VER_MAJOR__
					#if __CUDACC_VER_MAJOR__ < 9
					err += __shfl_down(err, tx);
					#else
					err += __shfl_down_sync(0xFFFFFFFF, err, tx);
					#endif
					#endif

				int warp_count = threadblock_size >> 5;
				if (warp_count > 1)
				{
					if (lane_id == 0)
						arr_sh[thread_id >> 5] = err;

					__syncthreads();

					if (thread_id < 32)
					{
						err = 0.0F;
						if (thread_id < warp_count)
							err = arr_sh[thread_id];
						#pragma unroll
						for(int tx = 4; tx > 0; tx >>= 1)
#ifdef __CUDACC_VER_MAJOR__
#if __CUDACC_VER_MAJOR__ < 9
							err += __shfl_down(err, tx);
#else
							err += __shfl_down_sync(0xFFFFFFFF, err, tx);
#endif
#endif
					}
				}
			}
		
			if (thread_id == 0)
				output[output_offset] = err * (mask * scale);
		}

		template<bool add_update_to_destination, int n_type>
		__global__ void mse_backprop_upd_kernel(
			float * __restrict output,
			const float * __restrict deriv_input_neurons,
			const float * __restrict target_input_neurons,
			float n_value,
			float n_value_m1,
			float scale2,
			int elem_count) 
		{
			int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
			if (elem_id < elem_count)
			{
				float gradient;
				float diff = target_input_neurons[elem_id] - deriv_input_neurons[elem_id];

				if (n_type == 1.0F)
				{
					gradient = (diff >= 0.0F ? 1.0F : -1.0F);
				}
				else if (n_type == 2.0F)
				{
					gradient = diff;
				}
				else
				{
					gradient = (diff >= 0.0F ? 1.0F : -1.0F) * __powf(fabsf(diff), n_value_m1);
				}

				if (add_update_to_destination)
					output[elem_id] += scale2 * gradient;
				else
					output[elem_id] = scale2 * gradient;
			}
		}

		template<bool add_update_to_destination, int n_type>
		__global__ void mse_backprop_upd_kernel(
			float * __restrict output,
			const float * __restrict deriv_input_neurons,
			const float * __restrict target_input_neurons,
			const float * __restrict scale_mask,
			float n_value,
			float n_value_m1,
			float scale2,
			int elem_count_per_feature_map,
			int input_feature_map_count,
			int entry_count) 
		{
			int neuron_id = blockDim.x * blockIdx.x + threadIdx.x;
			int feature_map_id = blockDim.y * blockIdx.y + threadIdx.y;
			int entry_id = blockDim.z * blockIdx.z + threadIdx.z;
			if ((neuron_id < elem_count_per_feature_map) && (feature_map_id < input_feature_map_count) && (entry_id < entry_count))
			{
				int elem_id = (entry_id * input_feature_map_count + feature_map_id) * elem_count_per_feature_map + neuron_id;
				float mask = scale_mask[entry_id * elem_count_per_feature_map + neuron_id];
				float gradient = 0.0F;
				if (mask != 0.0F)
				{
					float diff = target_input_neurons[elem_id] - deriv_input_neurons[elem_id];

					if (n_type == 1.0F)
					{
						gradient = (diff >= 0.0F ? 1.0F : -1.0F);
					}
					else if (n_type == 2.0F)
					{
						gradient = diff;
					}
					else
					{
						gradient = (diff >= 0.0F ? 1.0F : -1.0F) * __powf(fabsf(diff), n_value_m1);
					}
				}

				if (add_update_to_destination)
					output[elem_id] += gradient * (mask * scale2);
				else
					output[elem_id] = gradient * (mask * scale2);
			}
		}

		void lerror_layer_updater_cuda::enqueue_forward_propagation(
			hipStream_t stream_id,
			cuda_linear_buffer_device::ptr output_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data_custom,
			const std::vector<cuda_linear_buffer_device::const_ptr>& input_buffers,
			const std::vector<cuda_linear_buffer_device::const_ptr>& persistent_working_data,
			cuda_linear_buffer_device::ptr temporary_working_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_working_per_entry_buffer,
			cuda_linear_buffer_device::ptr temporary_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_per_entry_buffer,
			unsigned int entry_count)
		{
			int threadblock_size = get_threadblock_size(input_configuration_specific_list[0].feature_map_count);
			const float * scale_mask = 0;
			if (input_buffers.size() > 2)
				scale_mask = *input_buffers[2];

			int smem_size = ((threadblock_size + 32 - 1) / 32) * sizeof(float);
			if (n_value == 1.0F)
				lerror_upd_kernel<1><<<dim3(input_elem_count_per_feature_map_list[0], entry_count), threadblock_size, smem_size, stream_id>>>(
					*output_buffer,
					*input_buffers[0],
					*input_buffers[1],
					scale_mask,
					input_configuration_specific_list[0].feature_map_count,
					input_elem_count_per_feature_map_list[0],
					n_value,
					scale,
					entry_count);
			else if (n_value == 2.0F)
				lerror_upd_kernel<2><<<dim3(input_elem_count_per_feature_map_list[0], entry_count), threadblock_size, smem_size, stream_id>>>(
					*output_buffer,
					*input_buffers[0],
					*input_buffers[1],
					scale_mask,
					input_configuration_specific_list[0].feature_map_count,
					input_elem_count_per_feature_map_list[0],
					n_value,
					scale,
					entry_count);
			else
				lerror_upd_kernel<-1><<<dim3(input_elem_count_per_feature_map_list[0], entry_count), threadblock_size, smem_size, stream_id>>>(
					*output_buffer,
					*input_buffers[0],
					*input_buffers[1],
					scale_mask,
					input_configuration_specific_list[0].feature_map_count,
					input_elem_count_per_feature_map_list[0],
					n_value,
					scale,
					entry_count);
		}

		void lerror_layer_updater_cuda::enqueue_backward_data_propagation(
			hipStream_t stream_id,
			unsigned int input_index,
			cuda_linear_buffer_device::ptr input_errors_buffer,
			cuda_linear_buffer_device::const_ptr output_errors_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data_custom,
			const std::vector<cuda_linear_buffer_device::const_ptr>& input_neurons_buffers,
			cuda_linear_buffer_device::const_ptr output_neurons_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& persistent_working_data,
			cuda_linear_buffer_device::ptr temporary_working_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_working_per_entry_buffer,
			cuda_linear_buffer_device::const_ptr temporary_fixed_buffer,
			cuda_linear_buffer_device::const_ptr temporary_per_entry_buffer,
			bool add_update_to_destination,
			unsigned int entry_count)
		{
			if (input_neurons_buffers.size() > 2)
			{
				std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
					*cuda_config,
					input_elem_count_per_feature_map_list[0],
					input_configuration_specific_list[0].feature_map_count,
					entry_count);

				if (add_update_to_destination)
				{
					if (n_value == 1.0F)
						mse_backprop_upd_kernel<true, 1><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
							*input_errors_buffer,
							*input_neurons_buffers[input_index],
							*input_neurons_buffers[1 - input_index],
							*input_neurons_buffers[2],
							n_value,
							n_value - 1.0F,
							scale * n_value,
							input_elem_count_per_feature_map_list[0],
							input_configuration_specific_list[0].feature_map_count,
							entry_count);
					else if (n_value == 2.0F)
						mse_backprop_upd_kernel<true, 2><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
							*input_errors_buffer,
							*input_neurons_buffers[input_index],
							*input_neurons_buffers[1 - input_index],
							*input_neurons_buffers[2],
							n_value,
							n_value - 1.0F,
							scale * n_value,
							input_elem_count_per_feature_map_list[0],
							input_configuration_specific_list[0].feature_map_count,
							entry_count);
					else
						mse_backprop_upd_kernel<true, -1><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
							*input_errors_buffer,
							*input_neurons_buffers[input_index],
							*input_neurons_buffers[1 - input_index],
							*input_neurons_buffers[2],
							n_value,
							n_value - 1.0F,
							scale * n_value,
							input_elem_count_per_feature_map_list[0],
							input_configuration_specific_list[0].feature_map_count,
							entry_count);
				}
				else
				{
					if (n_value == 1.0F)
						mse_backprop_upd_kernel<false, 1><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
							*input_errors_buffer,
							*input_neurons_buffers[input_index],
							*input_neurons_buffers[1 - input_index],
							*input_neurons_buffers[2],
							n_value,
							n_value - 1.0F,
							scale * n_value,
							input_elem_count_per_feature_map_list[0],
							input_configuration_specific_list[0].feature_map_count,
							entry_count);
					else if (n_value == 2.0F)
						mse_backprop_upd_kernel<false, 2><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
							*input_errors_buffer,
							*input_neurons_buffers[input_index],
							*input_neurons_buffers[1 - input_index],
							*input_neurons_buffers[2],
							n_value,
							n_value - 1.0F,
							scale * n_value,
							input_elem_count_per_feature_map_list[0],
							input_configuration_specific_list[0].feature_map_count,
							entry_count);
					else
						mse_backprop_upd_kernel<false, -1><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
							*input_errors_buffer,
							*input_neurons_buffers[input_index],
							*input_neurons_buffers[1 - input_index],
							*input_neurons_buffers[2],
							n_value,
							n_value - 1.0F,
							scale * n_value,
							input_elem_count_per_feature_map_list[0],
							input_configuration_specific_list[0].feature_map_count,
							entry_count);
				}
			}
			else
			{
				int elem_count = entry_count * input_elem_count_per_entry_list[0];
				std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
					*cuda_config,
					elem_count);

				if (add_update_to_destination)
				{
					if (n_value == 1.0F)
						mse_backprop_upd_kernel<true, 1><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
							*input_errors_buffer,
							*input_neurons_buffers[input_index],
							*input_neurons_buffers[1 - input_index],
							n_value,
							n_value - 1.0F,
							scale * n_value,
							elem_count);
					else if (n_value == 2.0F)
						mse_backprop_upd_kernel<true, 2><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
							*input_errors_buffer,
							*input_neurons_buffers[input_index],
							*input_neurons_buffers[1 - input_index],
							n_value,
							n_value - 1.0F,
							scale * n_value,
							elem_count);
					else
						mse_backprop_upd_kernel<true, -1><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
							*input_errors_buffer,
							*input_neurons_buffers[input_index],
							*input_neurons_buffers[1 - input_index],
							n_value,
							n_value - 1.0F,
							scale * n_value,
							elem_count);
				}
				else
				{
					if (n_value == 1.0F)
						mse_backprop_upd_kernel<false, 1><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
							*input_errors_buffer,
							*input_neurons_buffers[input_index],
							*input_neurons_buffers[1 - input_index],
							n_value,
							n_value - 1.0F,
							scale * n_value,
							elem_count);
					else if (n_value == 2.0F)
						mse_backprop_upd_kernel<false, 2><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
							*input_errors_buffer,
							*input_neurons_buffers[input_index],
							*input_neurons_buffers[1 - input_index],
							n_value,
							n_value - 1.0F,
							scale * n_value,
							elem_count);
					else
						mse_backprop_upd_kernel<false, -1><<<kernel_dims.first, kernel_dims.second, 0, stream_id>>>(
							*input_errors_buffer,
							*input_neurons_buffers[input_index],
							*input_neurons_buffers[1 - input_index],
							n_value,
							n_value - 1.0F,
							scale * n_value,
							elem_count);
				}
			}
		}

		void lerror_layer_updater_cuda::updater_configured()
		{
			if (actions.find(layer_action(layer_action::backward_data, 2)) != actions.end())
				throw neural_network_exception("mse_layer_updater_cuda cannot do backward propagation for scale mask");

			std::shared_ptr<const lerror_layer> layer_derived = std::dynamic_pointer_cast<const lerror_layer>(layer_schema);

			scale = layer_derived->scale;
			n_value = layer_derived->n;
		}

		bool lerror_layer_updater_cuda::is_backward_data_dependent_on_output_buffer(unsigned int action_input_index) const
		{
			return false;
		}

		int lerror_layer_updater_cuda::get_threadblock_size(int input_feature_map_count)
		{
			int threadblock_size;

			if (input_feature_map_count < 256)
			{
				threadblock_size = (input_feature_map_count + 32 - 1) / 32 * 32;
			}
			else
			{
				int threadblock_count = (input_feature_map_count + 256 - 1) / 256;
				threadblock_size = (input_feature_map_count + threadblock_count - 1) / threadblock_count;
				threadblock_size = (threadblock_size + 32 - 1) / 32 * 32;
			}

			return threadblock_size;
		}
	}
}
