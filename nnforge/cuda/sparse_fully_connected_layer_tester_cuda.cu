#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2017 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "sparse_fully_connected_layer_tester_cuda.h"

#include <hip/hip_runtime.h>

#include "util_cuda.h"
#include "cudnn_util.h"
#include "neural_network_cudnn_exception.h"
#include "../sparse_convolution_layer.h"

namespace nnforge
{
	namespace cuda
	{
		#define OUTPUT_ELEM_COUNT_BLOCK_SIZE 4
		extern __shared__ float arr_sh[];
		__global__ void sparse_fully_connected_kernel(
			float * __restrict output_neurons,
			const float * __restrict input_neurons,
			const float * __restrict weights,
			const int * __restrict column_indices,
			const int * __restrict row_ptrs,
			int output_elem_count_per_entry,
			int input_elem_count_per_entry,
			int entry_count,
			int input_feature_map_block_size,
			int window_size)
		{
			int row_id = blockIdx.y * blockDim.y + threadIdx.y;
			if (row_id >= output_elem_count_per_entry)
				return;
			int start_column_index = __load_nc(row_ptrs + row_id);
			int end_column_index = __load_nc(row_ptrs + row_id + 1);
			int thread_id_x = blockIdx.x * blockDim.x + threadIdx.x;
			int base_column_index_offset = (thread_id_x >> 5) * input_feature_map_block_size;
			int base_nnz_index = start_column_index + base_column_index_offset;
			if (base_nnz_index >= end_column_index)
				return;

			int base_entry_id = (blockIdx.z * blockDim.z + threadIdx.z) * OUTPUT_ELEM_COUNT_BLOCK_SIZE;
			if (base_entry_id >= entry_count)
				return;

			int lane_id = thread_id_x & 31;
			int it_count = min(input_feature_map_block_size, end_column_index - base_nnz_index);

			int thread_id = blockDim.x * (threadIdx.z * blockDim.y + threadIdx.y) + threadIdx.x;
			int warp_id = thread_id >> 5;

			volatile int * column_indices_sh = (int *)arr_sh;
			if (lane_id < it_count)
				column_indices_sh[thread_id] = column_indices[base_nnz_index + lane_id];

			int window_it_count = (window_size + 31) >> 5;

			bool valid[OUTPUT_ELEM_COUNT_BLOCK_SIZE];
			int entry_ids[OUTPUT_ELEM_COUNT_BLOCK_SIZE];
			#pragma unroll
			for(int i = 0; i < OUTPUT_ELEM_COUNT_BLOCK_SIZE; ++i)
			{
				valid[i] = (i < (entry_count - base_entry_id));
				entry_ids[i] = valid[i] ? (base_entry_id + i) : (entry_count - 1);
			}

			float sums[OUTPUT_ELEM_COUNT_BLOCK_SIZE];
			#pragma unroll
			for(int i = 0; i < OUTPUT_ELEM_COUNT_BLOCK_SIZE; ++i)
				sums[i] = 0.0F;

			for(int i = 0; i < it_count; ++i)
			{
				int index = base_nnz_index + i;
				int column_id = column_indices_sh[warp_id * 32 + i];
				int local_weight_id = lane_id;
				for(int j = 0; j < window_it_count; ++j)
				{
					if (local_weight_id < window_size)
					{
						float w = __load_nc(weights + (int)(index * window_size + local_weight_id));
						#pragma unroll
						for(int k = 0; k < OUTPUT_ELEM_COUNT_BLOCK_SIZE; ++k)
						{
							float inp = __load_nc(input_neurons + entry_ids[k] * input_elem_count_per_entry + column_id * window_size + local_weight_id);
							sums[k] += w * inp;
						}
					}
					local_weight_id += 32;
				}
			}

			#pragma unroll
			for(int tx = 16; tx > 0; tx >>= 1)
			{
				#pragma unroll
				for(int i = 0; i < OUTPUT_ELEM_COUNT_BLOCK_SIZE; ++i)
#ifdef __CUDACC_VER_MAJOR__
#if __CUDACC_VER_MAJOR__ < 9
					sums[i] += __shfl_xor(sums[i], tx);
#else
					sums[i] += __shfl_xor_sync(0xFFFFFFFF, sums[i], tx);
#endif
#endif
			}

			if (lane_id == 0)
			{
				for(int i = 0; i < OUTPUT_ELEM_COUNT_BLOCK_SIZE; ++i)
					if (valid[i])
						atomicAdd(output_neurons + (base_entry_id + i) * output_elem_count_per_entry + row_id, sums[i]);
			}
		}

		const int sparse_fully_connected_layer_tester_cuda::max_input_feature_map_block_size = 32;

		sparse_fully_connected_layer_tester_cuda::sparse_fully_connected_layer_tester_cuda()
			: output_data_desc(0)
			, bias_desc(0)
		{
			cudnn_safe_call(hipdnnCreateTensorDescriptor(&output_data_desc));
			cudnn_safe_call(hipdnnCreateTensorDescriptor(&bias_desc));
		}

		sparse_fully_connected_layer_tester_cuda::~sparse_fully_connected_layer_tester_cuda()
		{
			hipdnnDestroyTensorDescriptor(output_data_desc);
			hipdnnDestroyTensorDescriptor(bias_desc);
		}

		void sparse_fully_connected_layer_tester_cuda::enqueue_forward_propagation(
			hipStream_t stream_id,
			cuda_linear_buffer_device::ptr output_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data_custom,
			const std::vector<cuda_linear_buffer_device::const_ptr>& input_buffers,
			const std::vector<cuda_linear_buffer_device::const_ptr>& persistent_working_data,
			cuda_linear_buffer_device::ptr temporary_working_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_working_per_entry_buffer,
			unsigned int entry_count)
		{
			cuda_util::set_with_value(
				*cuda_config,
				*output_buffer,
				0.0F,
				output_elem_count_per_entry * entry_count,
				stream_id);

			std::pair<int, int> input_feature_map_block_size_and_count = get_input_feature_map_block_size_and_count();
			std::pair<dim3, dim3> kernel_dims = cuda_util::get_grid_and_threadblock_sizes_sequential_access(
				*cuda_config,
				32 * input_feature_map_block_size_and_count.second,
				output_elem_count_per_entry,
				(entry_count + OUTPUT_ELEM_COUNT_BLOCK_SIZE - 1) / OUTPUT_ELEM_COUNT_BLOCK_SIZE,
				32);
			int threadblock_size = kernel_dims.second.x * kernel_dims.second.y * kernel_dims.second.z;
			int smem_size = threadblock_size * sizeof(float);
			sparse_fully_connected_kernel<<<kernel_dims.first, kernel_dims.second, smem_size, stream_id>>>(
				*output_buffer,
				*input_buffers[0],
				*data[0],
				*data_custom[0],
				*data_custom[1],
				output_elem_count_per_entry,
				input_elem_count_per_entry_list[0],
				entry_count,
				input_feature_map_block_size_and_count.first,
				window_size);

			// Add bias
			if (bias)
			{
				cudnn_safe_call(hipdnnSetStream(cuda_config->get_cudnn_handle(), stream_id));
				cudnn_util::set_tensor_descriptor(
					output_data_desc,
					output_configuration_specific,
					entry_count);
				float alpha = 1.0F;
				float beta = 1.0F;
				cudnn_safe_call(hipdnnAddTensor(
					cuda_config->get_cudnn_handle(),
					&alpha,
					bias_desc,
					*data[1],
					&beta,
					output_data_desc,
					*output_buffer));
			}
		}

		void sparse_fully_connected_layer_tester_cuda::tester_configured()
		{
			std::shared_ptr<const sparse_convolution_layer> layer_derived = std::dynamic_pointer_cast<const sparse_convolution_layer>(layer_schema);

			feature_map_connection_count = layer_derived->feature_map_connection_count;
			bias = layer_derived->bias;

			window_size = 1;
			for(std::vector<unsigned int>::const_iterator it = layer_derived->window_sizes.begin(); it != layer_derived->window_sizes.end(); ++it)
				window_size *= *it;

			cudnn_util::set_tensor_bias_descriptor(
				bias_desc,
				output_configuration_specific.feature_map_count,
				static_cast<unsigned int>(output_configuration_specific.dimension_sizes.size()));
		}

		void sparse_fully_connected_layer_tester_cuda::notify_data_custom(layer_data_custom::const_ptr host_data_custom)
		{
			max_column_index_count_per_row = 0;
			const std::vector<int>& row_indices = host_data_custom->at(1);
			for(int i = 0; i < row_indices.size() - 1; ++i)
				max_column_index_count_per_row = std::max(max_column_index_count_per_row, row_indices[i + 1] - row_indices[i]);
		}

		std::pair<int, int> sparse_fully_connected_layer_tester_cuda::get_input_feature_map_block_size_and_count() const
		{
			int candidate_block_size = max_column_index_count_per_row;

			if (candidate_block_size <= max_input_feature_map_block_size)
				return std::make_pair(candidate_block_size, 1);

			int candidate_block_count2 = (candidate_block_size + max_input_feature_map_block_size - 1) / max_input_feature_map_block_size;
			int candidate_block_size2 = (candidate_block_size + candidate_block_count2 - 1) / candidate_block_count2;

			return std::make_pair(candidate_block_size2, candidate_block_count2);
		}
	}
}
