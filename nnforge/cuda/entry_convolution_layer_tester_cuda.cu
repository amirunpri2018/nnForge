#include "hip/hip_runtime.h"
/*
 *  Copyright 2011-2016 Maxim Milakov
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "entry_convolution_layer_tester_cuda.h"

#include <hip/hip_runtime.h>

namespace nnforge
{
	namespace cuda
	{
		__global__ void entry_convolution_kernel(
			float * __restrict output,
			const float * __restrict input,
			int neuron_count_per_feature_map,
			int input_feature_map_count,
			int output_feature_map_count,
			int input_neuron_count,
			int output_neuron_count,
			int entry_count)
		{
			int neuron_id = blockIdx.x;
			int total_thread_id = blockIdx.y * blockDim.x + threadIdx.x;
			int feature_map_id = total_thread_id >> 5;
			int lane_id = total_thread_id & 31;
			int entry_id = blockIdx.z;

			if ((neuron_id < neuron_count_per_feature_map) && (feature_map_id < output_feature_map_count) && (entry_id < entry_count))
			{
				const float * in_base1 = input + entry_id * 2 * input_neuron_count + neuron_id;
				const float * in_base2 = in_base1 + input_neuron_count;

				int base_input_index1 = 0;
				int base_input_index2 = feature_map_id;
				if (feature_map_id > (input_feature_map_count - 1))
				{
					base_input_index1 = feature_map_id - (input_feature_map_count - 1);
					base_input_index2 = (input_feature_map_count - 1);
				}
				int iteration_count = min(input_feature_map_count - base_input_index1, base_input_index2 + 1);

				float sum = 0.0F;
				#pragma unroll 4
				for(int i = lane_id; i < iteration_count; i += 32)
					sum += in_base1[(base_input_index1 + i) * neuron_count_per_feature_map] * in_base2[(base_input_index2 - i) * neuron_count_per_feature_map];

				#pragma unroll
				for(int tx = 16; tx > 0; tx >>= 1)
#ifdef __CUDACC_VER_MAJOR__
#if __CUDACC_VER_MAJOR__ < 9
					sum += __shfl_down(sum, tx);
#else
					sum += __shfl_down_sync(0xFFFFFFFF, sum, tx);
#endif
#endif

				if (lane_id == 0)
					output[entry_id * output_neuron_count + feature_map_id * neuron_count_per_feature_map + neuron_id] = sum;
			}
		}

		void entry_convolution_layer_tester_cuda::enqueue_forward_propagation(
			hipStream_t stream_id,
			cuda_linear_buffer_device::ptr output_buffer,
			const std::vector<cuda_linear_buffer_device::const_ptr>& schema_data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data,
			const std::vector<cuda_linear_buffer_device::const_ptr>& data_custom,
			const std::vector<cuda_linear_buffer_device::const_ptr>& input_buffers,
			const std::vector<cuda_linear_buffer_device::const_ptr>& persistent_working_data,
			cuda_linear_buffer_device::ptr temporary_working_fixed_buffer,
			cuda_linear_buffer_device::ptr temporary_working_per_entry_buffer,
			unsigned int entry_count)
		{
			int warps_per_threadblock = 8;
			int threadblock_size = warps_per_threadblock * 32;
			int threadblocks_to_cover_all_feature_maps = (output_configuration_specific.feature_map_count + warps_per_threadblock - 1) / warps_per_threadblock;

			entry_convolution_kernel<<<dim3(output_elem_count_per_feature_map, threadblocks_to_cover_all_feature_maps, entry_count), threadblock_size, 0, stream_id>>>(
				*output_buffer,
				*input_buffers[0],
				output_elem_count_per_feature_map,
				input_configuration_specific_list[0].feature_map_count,
				output_configuration_specific.feature_map_count,
				input_elem_count_per_entry_list[0],
				output_elem_count_per_entry,
				entry_count);
		}
	}
}
